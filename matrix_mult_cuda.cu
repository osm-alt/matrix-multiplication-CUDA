#include "hip/hip_runtime.h"
%%cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int j, int k, int l) {
  
// Calculate the row index of the P element and M
int Row = blockIdx.y*blockDim.y+threadIdx.y;

// Calculate the column index of P and N
int Col = blockIdx.x*blockDim.x+threadIdx.x;

if ((Row < j) && (Col < l)) {
float Pvalue = 0;
// each thread computes one element of the block sub-matrix
for (int _k = 0; _k < k; ++_k) {
Pvalue += d_M[Row*k+_k]*d_N[_k*l+Col];
}

d_P[Row*l+Col] = Pvalue;
}
}


int main(int argc, char *argv[])
{
    clock_t t1 = clock();
    int j = 1000;
    int k = 500;
    int l = 800;
     float *h_M = (float*)malloc(j*k*sizeof(float));
     float *h_N = (float*)malloc(k*l*sizeof(float));
     float *h_P = (float*)malloc(j*l*sizeof(float));
 
    int i;
 
    for(i = 0; i < j * k; i++)
    {
        h_M[i] = 2;
    } 
  
    for(i = 0; i < k * l; i++)
    {
        h_N[i] = 3;
    }  
 
 
    int Msize = j * k * sizeof(float);
    int Nsize = k * l * sizeof(float);
    int Psize = j * l * sizeof(float);

    float *d_M, *d_N, *d_P;
    hipMalloc((void **) &d_M, Msize); 
    hipMalloc((void **) &d_N, Nsize);
    hipMalloc((void **) &d_P, Psize);

    hipMemcpy(d_M, h_M, Msize, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, Nsize, hipMemcpyHostToDevice);

    dim3 DimGrid((l-1)/32 + 1, (j-1)/32 + 1, 1);
    dim3 DimBlock(32, 32, 1);
 
    MatrixMulKernel<<<DimGrid, DimBlock>>>(d_M, d_N, d_P, j, k, l);
 hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s at line %d\n", hipGetErrorString(error), __LINE__);
        exit(-1);
    }

   hipMemcpy(h_P, d_P, Psize, hipMemcpyDeviceToHost);
 
   hipFree(d_M); hipFree(d_N); hipFree(d_P);
 
  free(h_M);
 free(h_N);
 free(h_P);

clock_t t2 = clock();
    printf("Elapsed time = % 5.3f seconds\n", (float)(t2 - t1) / CLOCKS_PER_SEC);
    return 0;
}